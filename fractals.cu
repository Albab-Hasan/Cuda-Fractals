#include <hip/hip_runtime.h>

#include <stdio.h>
#include <stdlib.h>
#include <math.h>

// image dimensions
#define WIDTH 1920
#define HEIGHT 1080
#define MAX_ITER 1000

// color structure
struct Color {
    unsigned char r, g, b;
};

// complex number operations
__device__ float2 complex_mul(float2 a, float2 b) {
    return make_float2(a.x * b.x - a.y * b.y, a.x * b.y + a.y * b.x);
}

__device__ float complex_mag_squared(float2 c) {
    return c.x * c.x + c.y * c.y;
}

// smooth coloring function
__device__ Color get_color(int iter, float smooth_iter) {
    if (iter == MAX_ITER) {
        return {0, 0, 0}; // black for points in the set
    }
    
    // create rainbow gradient with smooth transitions
    float t = (iter + smooth_iter) * 0.05f;
    float r = 0.5f + 0.5f * sinf(t);
    float g = 0.5f + 0.5f * sinf(t + 2.094f); // 2π/3
    float b = 0.5f + 0.5f * sinf(t + 4.188f); // 4π/3
    
    // add some brightness variation
    float brightness = 1.0f - expf(-0.1f * iter);
    
    return {
        (unsigned char)(255 * r * brightness),
        (unsigned char)(255 * g * brightness),
        (unsigned char)(255 * b * brightness)
    };
}

// CUDA kernel for mandelbrot set computation
__global__ void mandelbrot_kernel(Color* image, int width, int height, 
                                  double center_x, double center_y, double zoom) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    
    if (x >= width || y >= height) return;
    
    // map pixel coordinates to complex plane
    double scale = 4.0 / (zoom * min(width, height));
    double real = center_x + (x - width / 2.0) * scale;
    double imag = center_y + (y - height / 2.0) * scale;
    
    float2 c = make_float2(real, imag);
    float2 z = make_float2(0.0f, 0.0f);
    
    int iter = 0;
    float mag_sq = 0.0f;
    
    // mandelbrot iteration with early bailout
    while (iter < MAX_ITER && mag_sq < 16.0f) {
        z = complex_mul(z, z);
        z.x += c.x;
        z.y += c.y;
        mag_sq = complex_mag_squared(z);
        iter++;
    }
    
    // smooth iteration count for better coloring
    float smooth_iter = 0.0f;
    if (iter < MAX_ITER) {
        smooth_iter = logf(logf(sqrtf(mag_sq))) / logf(2.0f);
    }
    
    // generate color and store in image
    Color color = get_color(iter, smooth_iter);
    image[y * width + x] = color;
}

// julia set kernel for variety
__global__ void julia_kernel(Color* image, int width, int height,
                            double julia_real, double julia_imag, double zoom) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    
    if (x >= width || y >= height) return;
    
    double scale = 4.0 / (zoom * min(width, height));
    double real = (x - width / 2.0) * scale;
    double imag = (y - height / 2.0) * scale;
    
    float2 z = make_float2(real, imag);
    float2 c = make_float2(julia_real, julia_imag);
    
    int iter = 0;
    float mag_sq = 0.0f;
    
    while (iter < MAX_ITER && mag_sq < 16.0f) {
        z = complex_mul(z, z);
        z.x += c.x;
        z.y += c.y;
        mag_sq = complex_mag_squared(z);
        iter++;
    }
    
    float smooth_iter = 0.0f;
    if (iter < MAX_ITER) {
        smooth_iter = logf(logf(sqrtf(mag_sq))) / logf(2.0f);
    }
    
    Color color = get_color(iter, smooth_iter);
    image[y * width + x] = color;
}

// burning ship fractal kernel
__global__ void burning_ship_kernel(Color* image, int width, int height,
                                   double center_x, double center_y, double zoom) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    
    if (x >= width || y >= height) return;
    
    double scale = 4.0 / (zoom * min(width, height));
    double real = center_x + (x - width / 2.0) * scale;
    double imag = center_y - (y - height / 2.0) * scale; // Note: flipped for burning ship
    
    float2 c = make_float2(real, imag);
    float2 z = make_float2(0.0f, 0.0f);
    
    int iter = 0;
    float mag_sq = 0.0f;
    
    while (iter < MAX_ITER && mag_sq < 16.0f) {
        // Burning Ship: z = (|Re(z)| + i|Im(z)|)^2 + c
        z.x = fabsf(z.x);
        z.y = fabsf(z.y);
        z = complex_mul(z, z);
        z.x += c.x;
        z.y += c.y;
        mag_sq = complex_mag_squared(z);
        iter++;
    }
    
    float smooth_iter = 0.0f;
    if (iter < MAX_ITER) {
        smooth_iter = logf(logf(sqrtf(mag_sq))) / logf(2.0f);
    }
    
    Color color = get_color(iter, smooth_iter);
    image[y * width + x] = color;
}

// save image as PPM format
void save_ppm(const char* filename, Color* image, int width, int height) {
    FILE* fp = fopen(filename, "wb");
    if (!fp) {
        printf("Error: Could not open file %s\n", filename);
        return;
    }
    
    fprintf(fp, "P6\n%d %d\n255\n", width, height);
    fwrite(image, sizeof(Color), width * height, fp);
    fclose(fp);
    printf("Saved fractal to %s\n", filename);
}

// main function with fractal generation examples
int main() {
    // allocate memory
    Color* h_image = (Color*)malloc(WIDTH * HEIGHT * sizeof(Color));
    Color* d_image;
    
    hipMalloc(&d_image, WIDTH * HEIGHT * sizeof(Color));
    
    // CUDA grid configuration
    dim3 blockSize(16, 16);
    dim3 gridSize((WIDTH + blockSize.x - 1) / blockSize.x,
                  (HEIGHT + blockSize.y - 1) / blockSize.y);
    
    printf("Generating beautiful fractals with CUDA...\n");
    printf("Image size: %dx%d, Max iterations: %d\n", WIDTH, HEIGHT, MAX_ITER);
    
    // generate classic mandelbrot set
    printf("\n1. Generating Mandelbrot set...\n");
    mandelbrot_kernel<<<gridSize, blockSize>>>(d_image, WIDTH, HEIGHT, -0.5, 0.0, 1.0);
    hipDeviceSynchronize();
    hipMemcpy(h_image, d_image, WIDTH * HEIGHT * sizeof(Color), hipMemcpyDeviceToHost);
    save_ppm("mandelbrot_classic.ppm", h_image, WIDTH, HEIGHT);
    
    // generate zoomed mandelbrot
    printf("2. Generating zoomed Mandelbrot set...\n");
    mandelbrot_kernel<<<gridSize, blockSize>>>(d_image, WIDTH, HEIGHT, -0.7, 0.0, 100.0);
    hipDeviceSynchronize();
    hipMemcpy(h_image, d_image, WIDTH * HEIGHT * sizeof(Color), hipMemcpyDeviceToHost);
    save_ppm("mandelbrot_zoom.ppm", h_image, WIDTH, HEIGHT);
    
    // generate julia set variations
    printf("3. Generating Julia set (c = -0.7 + 0.27015i)...\n");
    julia_kernel<<<gridSize, blockSize>>>(d_image, WIDTH, HEIGHT, -0.7, 0.27015, 1.0);
    hipDeviceSynchronize();
    hipMemcpy(h_image, d_image, WIDTH * HEIGHT * sizeof(Color), hipMemcpyDeviceToHost);
    save_ppm("julia_1.ppm", h_image, WIDTH, HEIGHT);
    
    printf("4. Generating Julia set (c = -0.4 + 0.6i)...\n");
    julia_kernel<<<gridSize, blockSize>>>(d_image, WIDTH, HEIGHT, -0.4, 0.6, 1.0);
    hipDeviceSynchronize();
    hipMemcpy(h_image, d_image, WIDTH * HEIGHT * sizeof(Color), hipMemcpyDeviceToHost);
    save_ppm("julia_2.ppm", h_image, WIDTH, HEIGHT);
    
    printf("5. Generating Julia set (c = 0.285 + 0.01i)...\n");
    julia_kernel<<<gridSize, blockSize>>>(d_image, WIDTH, HEIGHT, 0.285, 0.01, 1.0);
    hipDeviceSynchronize();
    hipMemcpy(h_image, d_image, WIDTH * HEIGHT * sizeof(Color), hipMemcpyDeviceToHost);
    save_ppm("julia_3.ppm", h_image, WIDTH, HEIGHT);
    
    // generate burning ship fractal
    printf("6. Generating Burning Ship fractal...\n");
    burning_ship_kernel<<<gridSize, blockSize>>>(d_image, WIDTH, HEIGHT, -1.8, -0.08, 1.0);
    hipDeviceSynchronize();
    hipMemcpy(h_image, d_image, WIDTH * HEIGHT * sizeof(Color), hipMemcpyDeviceToHost);
    save_ppm("burning_ship.ppm", h_image, WIDTH, HEIGHT);
    
    // zoom into mandelbrot set
    printf("7. Generating deep zoom Mandelbrot...\n");
    mandelbrot_kernel<<<gridSize, blockSize>>>(d_image, WIDTH, HEIGHT, 
                                              -0.7269, 0.1889, 10000.0);
    hipDeviceSynchronize();
    hipMemcpy(h_image, d_image, WIDTH * HEIGHT * sizeof(Color), hipMemcpyDeviceToHost);
    save_ppm("mandelbrot_deep_zoom.ppm", h_image, WIDTH, HEIGHT);
    
    // cleanup
    free(h_image);
    hipFree(d_image);
    
    printf("\nAll fractals generated successfully!\n");
    
    return 0;
}
